#include "hip/hip_runtime.h"
#include "argmax_kernel.cuh"

__global__ void argmax_kernel(half* __restrict__ x, int size, int* result, volatile int* pPos, int* pPosGpu, bool write_token) {
    using BlockReduce = hipcub::BlockReduce<float, 1024>;
    __shared__ typename BlockReduce::TempStorage temp;
    __shared__ float shared_val;

    int tid = threadIdx.x;
    int step = blockDim.x;

    // find local max value and its position
    float max_val = tid < size ? (float)x[tid] : -INFINITY;
    int   max_pos = tid < size ? tid : 0;
    for (int i = tid + step; i < size; i += step) {
        if ((float)x[i] > max_val) {
            max_val = x[i];
            max_pos = i;
        }
    }

    // find the global max value
    float global_max_val;
    global_max_val = BlockReduce(temp).Reduce(max_val, hipcub::Max());
    if (threadIdx.x == 0)
        shared_val = global_max_val;
    __syncthreads();
    global_max_val = shared_val;

    // possibility of race condition here, so we first write it to shared memory variable and then have just one thread to update the pointers.
    __shared__ int global_max_pos;
    if (max_val == global_max_val) {
        global_max_pos = max_pos;
    }
    __syncthreads();

    // write next token to the current token location
    if (threadIdx.x == 0) {
        int token_pos = *pPos;
        token_pos++;

        if (write_token)
            result[token_pos] = global_max_pos;

        // update the token indices (unblocks the CPU)
        *pPos = token_pos;
        *pPosGpu = token_pos;
    }
}
