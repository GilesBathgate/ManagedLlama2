#include "hip/hip_runtime.h"
#include "softmax_kernel.cuh"

__global__ void softmax_kernel(half* __restrict__ arr, int num_heads, int* pPos) {
    __shared__ float att[MAX_SEQ_LEN_SMEM_KERNEL];
    int h = blockIdx.x;
    int tid = threadIdx.x;
    int step = blockDim.x;
    int size = *pPos + 1;

    // load input to shared memory
    for (int t = tid; t < size; t += step)
        att[t] = (float)arr[h * size + t];
    __syncthreads();

    using BlockReduce = hipcub::BlockReduce<float, 1024>;
    __shared__ typename BlockReduce::TempStorage temp;
    __shared__ float shared_val;

    // find max value (for numerical stability)
    float max_val = tid < size ? att[tid] : 0;
    for (int i = tid + step; i < size; i += step)
        if (att[i] > max_val)
            max_val = att[i];

    max_val = BlockReduce(temp).Reduce(max_val, hipcub::Max());
    if (threadIdx.x == 0)
        shared_val = max_val;
    __syncthreads();
    max_val = shared_val;

    // exp and sum
    float sum = 0.0f;
    for (int i = tid; i < size; i += step) {
        att[i] = expf(att[i] - max_val);
        sum += att[i];
    }

    sum = BlockReduce(temp).Sum(sum);
    if (threadIdx.x == 0)
        shared_val = sum;
    __syncthreads();
    sum = shared_val;

    // normalize and write the result
    for (int t = tid; t < size; t += step)
        arr[h * size + t] = (half)(att[t] / sum);
}
